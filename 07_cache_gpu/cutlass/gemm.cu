#include "hip/hip_runtime.h"
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#define DEBUG

#include <gemm/dispatch.h>
#include <gemm/epilogue_function.h>
//#include "util/matrix.h"
#include "util/timer.h"

using namespace cutlass;

// for Windows testing
float drand48() {
  return rand() / (RAND_MAX + 1.0);
}

int main(int argc, const char **argv) {
  // consts
  int m = 10240;
  int k = 4096;
  int n = 4096;
  float alpha = 1.0;
  float beta = 0.0;
  int g_timing_iterations = 10;

  // static const matrix_transform_t::kind_t TransformA = matrix_transform_t::NonTranspose;
  // static const matrix_transform_t::kind_t TransformB = matrix_transform_t::NonTranspose;
  static const int TransformA = 0;
  static const int TransformB = 0;

  // typedef float value_t;
  // typedef float accum_t;
  hipStream_t stream = 0;

  // definitions
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m*k * sizeof(float) );
  hipMallocManaged(&B, k*n * sizeof(float) );
  hipMallocManaged(&C, m*n * sizeof(float) );
  hipMallocManaged(&C2, m*n * sizeof(float) );
  // matrix<float> A(m, k);
  // matrix<float> B(k, n);
  // matrix<float> C(m, n);
  // matrix<float> C2(m, n);

  // fill out
  for( int jndex=0; jndex<k; jndex++ ) {
    for(int index=0; index<m; index++) {
      A[jndex*m + index] = drand48();
    }
  }
  for( int jndex=0; jndex<n; jndex++ ) {
    for(int index=0; index<k; index++) {
      B[jndex*k + index] = drand48();
    }
  }
  for( int jndex=0; jndex<n; jndex++ ) {
    for(int index=0; index<m; index++) {
      C[jndex*m + index] = 0;
      C2[jndex*m + index] = 0;
    }
  }
  hipDeviceSynchronize();  
  // A.random();
  // B.random();
  // C.fill_ramp(0,0);
  // C2.fill_ramp(0,0);
  // A.sync_device();
  // B.sync_device();
  // C.sync_device();
  // C2.sync_device();

  // CUBLAS
  hipblasHandle_t g_cublas_handle;
  hipblasCreate(&g_cublas_handle);
  gpu_timer timer;
  for (int i = 0; i < g_timing_iterations+2; i++) {
    if (i == 2) timer.start();
    // CUDA_PERROR(hipblasSgemm(
    //                         g_cublas_handle,
    //                         (hipblasOperation_t) TransformA,
    //                         (hipblasOperation_t) TransformB,
    //                         m,
    //                         n,
    //                         k,
    //                         &alpha,
    //                         A.d_data(),
    //                         m,
    //                         B.d_data(),
    //                         k,
    //                         &beta,
    //                         C.d_data(),
    //                         m));
    CUDA_PERROR(hipblasSgemm(
                            g_cublas_handle,
                            (hipblasOperation_t) TransformA,
                            (hipblasOperation_t) TransformB,
                            m,
                            n,
                            k,
                            &alpha,
                            A,
                            m,
                            B,
                            k,
                            &beta,
                            C,
                            m));
  }
  timer.stop();

  // calculate CUBLAS time
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = timer.elapsed_millis() / g_timing_iterations;
  double cublas_flops = double(num_flops) / tcublas / 1.0e6;

  // CUTLASS
  typedef gemm::blas_scaled_epilogue<float, float, float> epilogue_op_t;
  epilogue_op_t epilogue(alpha, beta);
  for (int i = 0; i < g_timing_iterations+2; i++) {
    if (i == 2) timer.start();
    // gemm::dispatch<epilogue_op_t>(
    //     m,
    //     n,
    //     k,
    //     alpha,
    //     beta,
    //     A.d_data(),
    //     B.d_data(),
    //     C2.d_data(),
    //     stream,
    //     false);
    gemm::dispatch<epilogue_op_t>(
        m,
        n,
        k,
        alpha,
        beta,
        A,
        B,
        C2,
        stream,
        false);
  }
  timer.stop();

  // calculate CUTLASS time
  double tcutlass = timer.elapsed_millis() / g_timing_iterations;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e6;

  // error performance summary. No need to optimize below this line
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  // C.sync_host();
  // C2.sync_host();
  hipDeviceSynchronize();  

  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      // err += fabs(C.get(i,j) - C2.get(i,j));
      err += fabs(C[i*m + j] - C2[i*m + j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipblasDestroy(g_cublas_handle);
}
